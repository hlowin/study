#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include <librealsense2/rs.hpp>
#include <opencv2/opencv.hpp>

__global__
void GrayscaleKernel(
  uchar* img_src,
  uchar* img_dst,
	const int width,
	const int height
) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int x = i % width;
  const int y = i / width;

  const int img_pos = x * 3 + y * 3 * width;
  const uchar img_gs = (img_src[img_pos + 0] + img_src[img_pos + 1] + img_src[img_pos + 2]) / 3;
  img_dst[img_pos + 0] = img_gs;
  img_dst[img_pos + 1] = img_gs;
  img_dst[img_pos + 2] = img_gs;
}

__global__
void CopyKernel(
  uchar* img_src,
  uchar* img_dst,
	const int width,
	const int height
) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int x = i % width;
  const int y = i / width;

  const int img_pos = x * 3 + y * 3 * width;
  img_dst[img_pos + 0] = img_src[img_pos + 0];
  img_dst[img_pos + 1] = img_src[img_pos + 1];
  img_dst[img_pos + 2] = img_src[img_pos + 2];
}

int main() {
	rs2::config cfg;
	const int width = 640;
	const int height = 480;
	const int array_size = width * height;
	cfg.enable_stream(RS2_STREAM_COLOR, width, height, RS2_FORMAT_BGR8, 30);

	rs2::pipeline pipe;
	pipe.start(cfg);

	cv::Mat color(cv::Size(width, height), CV_8UC3);
	// cv::Mat color = cv::imread("../exit.png");
	
	cv::Size size = color.size();

	cv::Mat img_result(size, CV_8UC3);

	uchar *img_color_device;
	hipMalloc(&img_color_device, array_size * 3 * sizeof(uchar));

	uchar *img_result_device;
	hipMalloc(&img_result_device, array_size * 3 * sizeof(uchar));

	const int thread = 128;
	const int block = (size.width * size.height) / thread;

	while (cv::waitKey(1) != 'q') {
		rs2::frameset frames = pipe.wait_for_frames();
		rs2::frame color_frame = frames.get_color_frame();
		std::memcpy(color.data, color_frame.get_data(), array_size * 3 * sizeof(uchar));

		hipMemcpy(img_color_device, color.data, array_size * 3 * sizeof(uchar), hipMemcpyHostToDevice);
		GrayscaleKernel<<<block, thread>>>(img_color_device, img_result_device, size.width, size.height);
    hipMemcpy(img_result.data, img_result_device, array_size * 3 * sizeof(uchar), hipMemcpyDeviceToHost);

		cv::namedWindow("color", cv::WINDOW_AUTOSIZE);
		cv::imshow("color", color);

		cv::namedWindow("result", cv::WINDOW_AUTOSIZE);
		cv::imshow("result", img_result);
	}
	cv::destroyAllWindows();
	hipFree(img_color_device);
	hipFree(img_result_device);

	return 0;
}
