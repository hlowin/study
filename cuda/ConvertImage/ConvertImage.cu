#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include <opencv2/opencv.hpp>

__global__
void GrayscaleKernel(uchar* img_src, uchar* img_dst, const int width,
                                const int height) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int x = i % width;
  const int y = i / width;

  const int img_pos = x * 3 + y * 3 * width;
  const uchar img_gs =
      (img_src[img_pos + 0] + img_src[img_pos + 1] + img_src[img_pos + 2]) / 3;
  img_dst[img_pos + 0] = img_gs;
  img_dst[img_pos + 1] = img_gs;
  img_dst[img_pos + 2] = img_gs;
}

__global__
void CopyKernel(uchar* img_src, uchar* img_dst, const int width,
                           const int height) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int x = i % width;
  const int y = i / width;

  const int img_pos = x * 3 + y * 3 * width;
  img_dst[img_pos + 0] = img_src[img_pos + 0];
  img_dst[img_pos + 1] = img_src[img_pos + 1];
  img_dst[img_pos + 2] = img_src[img_pos + 2];
}

int main() {
  cv::Mat color = cv::imread("../exit.png");
  cv::Size size = color.size();
  cv::Mat img_result(size, CV_8UC3);

  const int width = size.width;
  const int height = size.height;
  const int array_size = width * height;

  uchar* img_color_device;
  hipMalloc(&img_color_device, array_size * 3 * sizeof(uchar));

  uchar* img_result_device;
  hipMalloc(&img_result_device, array_size * 3 * sizeof(uchar));

  const int thread = 128;
  const int block = (size.width * size.height) / thread;

  while (cv::waitKey(1) != 'q') {
    hipMemcpy(img_color_device, color.data, array_size * 3 * sizeof(uchar),
               hipMemcpyHostToDevice);
    GrayscaleKernel<<<block, thread>>>(img_color_device, img_result_device,
                                       size.width, size.height);
    hipMemcpy(img_result.data, img_result_device,
               array_size * 3 * sizeof(uchar), hipMemcpyDeviceToHost);

    cv::namedWindow("color", cv::WINDOW_AUTOSIZE);
    cv::imshow("color", color);

    cv::namedWindow("result", cv::WINDOW_AUTOSIZE);
    cv::imshow("result", img_result);
  }
  cv::destroyAllWindows();
  hipFree(img_color_device);
  hipFree(img_result_device);

  return 0;
}
