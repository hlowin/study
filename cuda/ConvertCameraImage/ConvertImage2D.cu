#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include <librealsense2/rs.hpp>
#include <opencv2/opencv.hpp>

__global__
void GrayscaleKernel(
	uchar3* img_src,
	const size_t src_pitch,
	uchar3* result,
	const size_t dst_pitch,
	const int width,
	const int height
) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < width && y < height) {
		int i = x + y * src_pitch / sizeof(uchar3);
		const uchar3 col = img_src[i];
		const uchar gs = (col.x + col.y + col.z) / 3;
		i = x + y * dst_pitch / sizeof(uchar3);
		result[i] = {gs, gs, gs};
	}
}

__global__
void CopyKernel(
	uchar3* img_src,
	const size_t src_pitch,
	uchar3* result,
	const size_t dst_pitch,
	const int width,
	const int height
) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < width && y < height) {
		const int i = x + y * src_pitch / sizeof(uchar3);
		const uchar3 col = img_src[i];
		result[i] = {col.x, col.y, col.z};
	}
}

int main() {
	rs2::config cfg;
	const int width = 640;
	const int height = 480;
	cfg.enable_stream(RS2_STREAM_COLOR, width, height, RS2_FORMAT_BGR8, 30);

	rs2::pipeline pipe;
	pipe.start(cfg);

	cv::Mat color(cv::Size(width, height), CV_8UC3);
	// cv::Mat color = cv::imread("../exit.png");
	
	cv::Size size = color.size();

	cv::Mat img_result(size, CV_8UC3);

	uchar3 *img_color_device;
	size_t color_pitch_device;
	hipMallocPitch(&img_color_device, &color_pitch_device, size.width * sizeof(uchar3), size.height);

	uchar3 *img_result_device;
	size_t result_pitch_device;
	hipMallocPitch(&img_result_device, &result_pitch_device, size.width * sizeof(uchar3), size.height);

	// dim3 dimBlock(32, 32);
	dim3 dimBlock(16, 16);
	dim3 dimGrid(size.width / dimBlock.x, size.height / dimBlock.y);

	while (cv::waitKey(1) != 'q') {
		rs2::frameset frames = pipe.wait_for_frames();
		rs2::frame color_frame = frames.get_color_frame();
		std::memcpy(color.data, color_frame.get_data(), width * height * sizeof(uchar3));

		hipMemcpy2D(img_color_device, color_pitch_device, color.data, color.step,
			size.width * sizeof(uchar3), size.height, hipMemcpyDefault);
		CopyKernel<<<dimGrid, dimBlock>>>(img_color_device, color_pitch_device,
			img_result_device, result_pitch_device, size.width, size.height);
		hipMemcpy2D(img_result.data, img_result.step, img_result_device, result_pitch_device,
			size.width * sizeof(uchar3), size.height, hipMemcpyDefault);

		cv::namedWindow("color", cv::WINDOW_AUTOSIZE);
		cv::imshow("color", color);

		cv::namedWindow("result", cv::WINDOW_AUTOSIZE);
		cv::imshow("result", img_result);
	}
	cv::destroyAllWindows();
	hipFree(img_color_device);
	hipFree(img_result_device);

	return 0;
}
